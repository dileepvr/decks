#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <errno.h>
#include <string.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "main.h"
extern "C" {
#include "fileio.h"
}


__global__ void mykernel() {

}

bool debug_trace = true, psoft = false, dsoft = false;
bool record_allbank = false;
int ndecks, ntrials, houserules, strategy, nbets;
float penet, bank, startbank, minbet, betspread;
char mystring[64];
int shoe[52*8], dealer[21], player[8][21];
float bets[8];
int ncards, maxcardpos, cardpos = 0;
int handno, cardno, nhands, ptotal[8], paces[8], dtotal, daces, dcardno;
float *allbank;

int main(int argc, char* argv[]) {

  int i;
  int seed = time(NULL);  
  srand(seed);

  read_params(argv[1]);
  initialize_shoe(shoe, ndecks);
  shuffle(shoe, ndecks);

  ncards = 52*8;
  maxcardpos = floor(ncards*penet);


  if(record_allbank) {
    allbank = (float*)malloc(sizeof(float)*ntrials*nbets);
  }
  
  startbank = bank;
  if(debug_trace) { printf("Playing trials...."); }
  for(i = 0; i < ntrials; i++) {
    play(i);
  }
  if(debug_trace) { printf(" done.\n"); }  
 
  if(record_allbank) {

    sprintf(mystring,argv[1]);
    strcat(mystring,".allbank.dat");
    if(debug_trace) {
      printf("Writing allbank to %s ....", mystring);
    }    
    write_allbank(mystring, allbank, ntrials, nbets);
    if(debug_trace) { printf(" done.\n"); }    
    free(allbank);

  }
 
}

void play(int trialnum) {
  int curbets = 1, playeraction, flag = 0;
  bank = startbank;
  while ( !trialover(curbets) ) {
    //    curbets++;
    handno = 0, nhands = 1;    
    opendraw();
    while( flag == 0 ) {
      playeraction = verb(1);
      switch(playeraction) {
      case 0: // surrender
	bank += bets[handno--]/2.0;
	if (handno < 0) { flag = 1; } // Last hand surrendered
	else { recomputeptotal(); }
	break; 
      case 1: // stand
	if (--handno < 0) { flag = 2; } // Last hand stood
	else { recomputeptotal(); }	
	break;
      case 2: // hit
	hitdeal();
	if (handbust()) {
	  if (--handno < 0) { flag = 3; } // Last hand busted
	  else { recomputeptotal(); }	  
	}
	break;
      case 3: // double
	bank -= bets[handno];
	bets[handno] = 2*bets[handno];
	hitdeal();
	if (--handno < 0) { flag = 4; } // Last hand doubled
	else { recomputeptotal(); }	
	break;
      case 4: // split
	splitdeal(); // Not checking if split command is valid
	break;
      default: // do nothing
	break;
      }
    }
    resolvedeal();
    if(record_allbank) {
      allbank[nbets*trialnum+curbets-1] = bank;
    }
    curbets++;
    //    printhands();
  }
}

void printhands() {
  int ll, temp;

  printf("Dealer: ");
  for(ll = 0; ll < dcardno; ll++) {
    printf("%d ", dealer[ll]);
  }
  printf("\ndtotal = %d ", dtotal);
  if (dtotal > 21) { printf("(BUST!)\n"); }
  else { printf("\n"); }

  for(temp = 0; temp < nhands; temp++) {
    printf("Player hand %d: ", temp);
    ll = 0;
    while(player[temp][ll] != 0) {
      printf("%d ", player[temp][ll++]);
    } printf("\nptotal = %d ",ptotal[temp]);
    if(ptotal[temp] > 21){
      printf("(BUST!)\n");
    } else if (ptotal[temp] == dtotal) {
      printf("(PUSH!)\n");      
    } else if((ptotal[temp] > dtotal) || (dtotal > 21)) {
      printf("(WIN!)\n");            
    } else {
      printf("(LOSS!)\n");                  
    }
  }
}

void resolvedeal() {

  // Surrender not being accounted for right now
  bool allbusted = true;
  dcardno = 2;
  // Check player busts and finish dealer card draws
  for(handno = 0; handno < nhands; handno++) {
    recomputeptotal();
    if(ptotal[handno] <= 21) { allbusted = false; }
  }

  if(!allbusted) {
    if(dealer[0] == 1) {
      dtotal += 11; dsoft = true; daces++;
    } else if (dealer[0] > 10) {
      dtotal += 10;
    } else { dtotal += dealer[0]; }

    if(dtotal > 21) {
      while(daces > 0) {
	dtotal -= 10;
	if(dtotal <= 21) { daces--; break; }
      }
    }

    // Modify soft-17 rule based on 'houserules' parameter
    while((dtotal < 17) || ((dtotal == 17) && (daces > 0))) {
      if( cardpos >= maxcardpos ) {
	shuffle(shoe, ndecks);
	cardpos = 0;
      }
      dealer[dcardno++] = shoe[cardpos++];
      updatedtotal();
    }

    for(handno = 0; handno < nhands; handno++) {
      if((ptotal[handno] <= 21) && ((ptotal[handno] > dtotal) || (dtotal > 21))) {
	// Check for Blackjack
	// Modify this according to 'houserules' parameter
	if((ptotal[handno] == 21) && (player[handno][2] == 0)) {
	  bank += 2.5*bets[handno];
	} else {
	  bank += 2*bets[handno];
	}
      }

    }
  }
  
}

void splitdeal() {
  nhands++;
  if( cardpos >= maxcardpos - 2 ) {
    shuffle(shoe, ndecks);
    cardpos = 0;
  }
  bank -= bets[handno++];
  bets[handno] = bets[handno-1];
  player[handno+1][0] = player[handno++][1];
  player[handno-1][1] = shoe[cardpos++];
  player[handno][1] = shoe[cardpos++];
  cardno = 1;

  if (player[handno][0] == 1) {
    ptotal[handno] = 11; psoft = true; paces[handno]++;
  } else if (player[handno][0] > 10) {
    ptotal[handno] = 10;
  } else { ptotal[handno] = player[handno][0]; }

  if (player[handno][1] == 1) {
    ptotal[handno] += 11; psoft = true; paces[handno]++;
  } else if (player[handno][1] > 10) {
    ptotal[handno] += 10;
  } else { ptotal[handno] += player[handno][1]; }
  
}

void hitdeal() {
  if( cardpos >= maxcardpos ) {
    shuffle(shoe, ndecks);
    cardpos = 0;
  }
  player[handno][cardno++] = shoe[cardpos++];
  updateptotal();
}

void recomputeptotal() {

  cardno = 0; ptotal[handno] = 0; paces[handno] = 0;
  while( player[handno][cardno] != 0 ) {
    if (player[handno][cardno] == 1) {
      ptotal[handno] += 11; psoft = true;
      paces[handno]++;
    } else if (player[handno][cardno] > 10) {
      ptotal[handno] += 10;
    } else { ptotal[handno] += player[handno][cardno]; }
    cardno++;
  }
  //  cardno--;
  if (ptotal[handno] > 21) {
    while(paces[handno] > 0) {
      ptotal[handno] -= 10; paces[handno]--;
      if (ptotal[handno] <= 21) { break; }
    }
  }
  if (paces[handno] == 0) { psoft = false; }

}

void updateptotal() {

  if (player[handno][cardno-1] == 1) {
    ptotal[handno] += 11; psoft = true; paces[handno]++;
  } else if (player[handno][cardno-1] > 10) {
    ptotal[handno] += 10;
  } else { ptotal[handno] += player[handno][cardno-1]; }

  if (ptotal[handno] > 21 ) {
    /*    for( int ll = 0; ll < cardno; ll++) {
	  if (player[handno][ll] == 1) { paces[handno]++; }
	  }
    */
    while(paces[handno] > 0) {
      ptotal[handno] -= 10; paces[handno]--;
      if (ptotal[handno] < 21) { break; }
    }
    if (paces[handno] == 0) { psoft = false; }
  }
  
}

void updatedtotal() {

  if (dealer[dcardno-1] == 1) {
    dtotal += 11; dsoft = true; daces++;
  } else if (dealer[dcardno-1] > 10) {
    dtotal += 10;
  } else { dtotal += dealer[dcardno-1]; }

  if (dtotal > 21 ) {
    /*    for( int ll = 0; ll < dcardno; ll++) {
	  if (dealer[ll] == 1) { daces++; }
	  }
    */
    while(daces > 0) {
      dtotal -= 10; daces--;
      if (dtotal< 21) { break; }
    }
    if (daces == 0) { dsoft = false; }
  }
  
}

void opendraw() {
  cleartable();
  if( cardpos >= maxcardpos - 4 ) {
    shuffle(shoe, ndecks);
    cardpos = 0;
  }
  bets[handno] = verb(0);
  bank -= bets[handno];
  // dealer[0] card is hidden from player
  dealer[0] = shoe[cardpos++];
  dealer[1] = shoe[cardpos++];
  player[handno][0] = shoe[cardpos++];
  player[handno][1] = shoe[cardpos++];
  cardno = 2;

  if (dealer[1] == 1) {
    dtotal = 11; dsoft = true; daces++;
  } else if (dealer[1] > 10) {
    dtotal = 10;
  } else { dtotal = dealer[1]; }
  
  if (player[handno][0] == 1) {
    ptotal[handno] = 11; psoft = true; paces[handno]++;
  } else if (player[handno][0] > 10) {
    ptotal[handno] = 10;
  } else { ptotal[handno] = player[handno][0]; }

  if (player[handno][1] == 1) {
    ptotal[handno] += 11; psoft = true; paces[handno]++;
  } else if (player[handno][1] > 10) {
    ptotal[handno] += 10;
  } else { ptotal[handno] += player[handno][1]; }
  
}

bool handbust() {
  if (ptotal[handno] > 21 && paces[handno] == 0) {
    return true;
  }
  return false;
}

int verb(int state) {
  // This is the player action function
  switch(state) {
  case 0: // pre-deal betting cycle, return bet amount
    return openbet();
  case 1: // cards dealt, return action
    return pdecision();
  default: // Same as case 0
    return openbet();
  }
}

int openbet() {
  switch(strategy) {
  case 0: // Test strategy
    return minbet;
  default: // Same as case 0
    return minbet;
  }
}

int pdecision() {
  /* return meanings
     0 == surrender
     1 == stand
     2 == hit
     3 == double
     4 == split
  */

  switch(strategy) {
  case 0: // Test strategy, hit if ptotal < 17, else stand
  default: // Same as case 0
    if (ptotal[handno] < 17) {
      return 2;
    } else {
      return 1;
    }
  }
 
}

void cleartable() {
  // Clean up hands
  for(int i = 0; i < 21; i++) {
    dealer[i] = 0;
    for(int j = 0; j < 8; j++) { player[j][i] = 0; }
  }
  for(int k = 0; k < 8; k++) {
    bets[k] = 0.0; ptotal[k] = 0; paces[k] = 0;
  }
  dtotal = 0; psoft = false; dsoft = false; daces = 0;
}

bool trialover(int curbetno) {
  if (nbets > 0) {
    if (curbetno > nbets || bank <= 0) {
      return true; } else {return false;}
  } else if (nbets < 0) {
    if (curbetno > abs(nbets)) { return true; }
    else { return false; }
  } else {
    if (bank <= 0) { return true; } else { return false; }
  }
}

void read_params(char* fname) {

  sprintf(mystring,"debug_trace");
  get_bool_param(fname, mystring, &debug_trace, true);
  sprintf(mystring,"ndecks");
  get_int_param(fname, mystring, &ndecks, debug_trace);
  if (ndecks < 1) { ndecks = 1; }
  if (ndecks > 8) { ndecks = 8; }
  sprintf(mystring,"ntrials");  
  get_int_param(fname, mystring, &ntrials, debug_trace);
  sprintf(mystring,"houserules");  
  get_int_param(fname, mystring, &houserules, debug_trace);
  sprintf(mystring,"strategy");  
  get_int_param(fname, mystring, &strategy, debug_trace);
  sprintf(mystring,"nbets");  
  get_int_param(fname, mystring, &nbets, debug_trace);
  sprintf(mystring,"penetration");  
  get_real_param(fname, mystring, &penet, debug_trace);
  if (penet < 2.0) { penet = 10.0; }
  if (penet > 100.0) { penet = 100.0; }  
  sprintf(mystring,"bank");  
  get_real_param(fname, mystring, &bank, debug_trace);
  sprintf(mystring,"minbet");  
  get_real_param(fname, mystring, &minbet, debug_trace);
  if (minbet > bank) { minbet = bank; }
  sprintf(mystring,"betspread");  
  get_real_param(fname, mystring, &betspread, debug_trace);
  sprintf(mystring,"record_allbank");
  get_bool_param(fname, mystring, &record_allbank, debug_trace);
  
}

void swap(int *a, int *b) {
  int temp = *a;
  *a = *b;
  *b = temp;
}


// Fischer-Yates random permutation of arr[]
// Move this to GPU eventually
void shuffle( int* arr, int ndeck ) {

  int ii, jj, kk;
  //  srand ( time(NULL) );

  // Shuffle thrice, just 'cause
  for (kk = 0; kk < 3; kk++) {
    // Start from the last element and swap one by one. We don't
    // need to run for the first element that's why ii > 0
    for (ii = ndeck*52-1; ii > 0; ii--){

      // Pick a random index from 0 to ii
      jj = rand() % (ii+1);

      // Swap arr[i] with the element at random index
      swap(&arr[ii], &arr[jj]);
    }
  }
}

void initialize_shoe(int* arr, int ndeck) {

  // 1s are aces, J = 11, Q = 12, K = 13
  for (int ii = 0; ii < ndeck; ii++) {
    for (int jj = 0; jj < 4; jj++) {    
      for (int kk = 0; kk < 13; kk++) {
	arr[ii*52+jj*13+kk] = kk+1;
      }
    }
  }

}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <errno.h>
#include <string.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "main.h"
extern "C" {
#include "fileio.h"
}


__global__ void mykernel() {

}

bool debug_trace = true;
int ndecks, ntrials, houserules, strategy;
float penet, bank, minbet, betspread;
char mystring[64];
int shoe[52*8]; 

int main(int argc, char* argv[]) {

  int i;

  int seed = time(NULL);  
  srand(seed);

  read_params(argv[1]);
  initialize_shoe(shoe, ndecks);
  shuffle(shoe, ndecks);

  for(i = 0; i < ndecks*13; i++) {
    printf("shoe[%d] = %d\n", i, shoe[i]);
  }
  
}


void read_params(char* fname) {

  sprintf(mystring,"debug_trace");
  get_bool_param(fname, mystring, &debug_trace, true);
  sprintf(mystring,"ndecks");
  get_int_param(fname, mystring, &ndecks, debug_trace);
  if (ndecks < 1) { ndecks = 1; }
  if (ndecks > 8) { ndecks = 8; }
  sprintf(mystring,"ntrials");  
  get_int_param(fname, mystring, &ntrials, debug_trace);
  sprintf(mystring,"houserules");  
  get_int_param(fname, mystring, &houserules, debug_trace);
  sprintf(mystring,"strategy");  
  get_int_param(fname, mystring, &strategy, debug_trace);      
  sprintf(mystring,"penetration");  
  get_real_param(fname, mystring, &penet, debug_trace);
  if (penet < 0.0) { penet = 10.0; }
  if (penet > 100.0) { penet = 100.0; }  
  sprintf(mystring,"bank");  
  get_real_param(fname, mystring, &bank, debug_trace);
  sprintf(mystring,"minbet");  
  get_real_param(fname, mystring, &minbet, debug_trace);
  if (minbet > bank) { minbet = bank; }
  sprintf(mystring,"betspread");  
  get_real_param(fname, mystring, &betspread, debug_trace);  
  
}

void swap(int *a, int *b) {
  int temp = *a;
  *a = *b;
  *b = temp;
}


// Fischer-Yates random permutation of arr[]
// Move this to GPU eventually
void shuffle( int* arr, int ndeck ) {

  int ii, jj, kk;
  //  srand ( time(NULL) );

  // Shuffle thrice, just 'cause
  for (kk = 0; kk < 3; kk++) {
    // Start from the last element and swap one by one. We don't
    // need to run for the first element that's why ii > 0
    for (ii = ndeck*13-1; ii > 0; ii--){

      // Pick a random index from 0 to ii
      jj = rand() % (ii+1);

      // Swap arr[i] with the element at random index
      swap(&arr[ii], &arr[jj]);
    }
  }
}

void initialize_shoe(int* arr, int ndeck) {

  // 1s are aces
  for (int ii = 0; ii < ndeck; ii++) {
    for (int kk = 0; kk < 10; kk++) {
      arr[ii*10+kk] = kk+1;
    }
  }
  // Fill up remaining face-10s
  for (int ll = 0; ll < 3*ndeck; ll++) {
    arr[ndeck*10+ll] = 10;
  }

}

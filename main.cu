#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <errno.h>
#include <string.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "main.h"
extern "C" {
#include "fileio.h"
}


__global__ void mykernel() {

}

bool debug_trace = true;
int ndecks, ntrials, houserules, strategy;
float penet, bank, minbet, betspread;
char mystring[64];
int shoe[52*8]; 

int main(int argc, char* argv[]) {

  int i;

  int seed = time(NULL);  
  srand(seed);

  read_params(argv[1]);
  initialize_shoe(shoe, ndecks);
  shuffle(shoe, ndecks);

  for(i = 0; i < ndecks*52; i++) {
    printf("shoe[%d] = %d\n", i, shoe[i]);
  }
  
}


void read_params(char* fname) {

  sprintf(mystring,"debug_trace");
  get_bool_param(fname, mystring, &debug_trace, true);
  sprintf(mystring,"ndecks");
  get_int_param(fname, mystring, &ndecks, debug_trace);
  if (ndecks < 1) { ndecks = 1; }
  if (ndecks > 8) { ndecks = 8; }
  sprintf(mystring,"ntrials");  
  get_int_param(fname, mystring, &ntrials, debug_trace);
  sprintf(mystring,"houserules");  
  get_int_param(fname, mystring, &houserules, debug_trace);
  sprintf(mystring,"strategy");  
  get_int_param(fname, mystring, &strategy, debug_trace);      
  sprintf(mystring,"penetration");  
  get_real_param(fname, mystring, &penet, debug_trace);
  if (penet < 0.0) { penet = 10.0; }
  if (penet > 100.0) { penet = 100.0; }  
  sprintf(mystring,"bank");  
  get_real_param(fname, mystring, &bank, debug_trace);
  sprintf(mystring,"minbet");  
  get_real_param(fname, mystring, &minbet, debug_trace);
  if (minbet > bank) { minbet = bank; }
  sprintf(mystring,"betspread");  
  get_real_param(fname, mystring, &betspread, debug_trace);  
  
}

void swap(int *a, int *b) {
  int temp = *a;
  *a = *b;
  *b = temp;
}


// Fischer-Yates random permutation of arr[]
// Move this to GPU eventually
void shuffle( int* arr, int ndeck ) {

  int ii, jj, kk;
  //  srand ( time(NULL) );

  // Shuffle thrice, just 'cause
  for (kk = 0; kk < 3; kk++) {
    // Start from the last element and swap one by one. We don't
    // need to run for the first element that's why ii > 0
    for (ii = ndeck*52-1; ii > 0; ii--){

      // Pick a random index from 0 to ii
      jj = rand() % (ii+1);

      // Swap arr[i] with the element at random index
      swap(&arr[ii], &arr[jj]);
    }
  }
}

void initialize_shoe(int* arr, int ndeck) {

  // 1s are aces, J = 11, Q = 12, K = 13
  for (int ii = 0; ii < ndeck; ii++) {
    for (int jj = 0; jj < 4; jj++) {    
      for (int kk = 0; kk < 13; kk++) {
	arr[ii*52+jj*13+kk] = kk+1;
      }
    }
  }


}

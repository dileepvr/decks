#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <errno.h>
#include <string.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "main.h"
extern "C" {
#include "fileio.h"
}


__global__ void mykernel() {

}

bool debug_trace = true, psoft = false, dsoft = false;
int ndecks, ntrials, houserules, strategy, nbets;
float penet, bank, startbank, minbet, betspread;
char mystring[64];
int shoe[52*8], dealer[21], player[8][21];
float bets[8];
int ncards, maxcardpos, cardpos = 0;
int handno, cardno, nhands, ptotal[8], paces[8], dtotal, daces;

int main(int argc, char* argv[]) {

  int i;
  int seed = time(NULL);  
  srand(seed);

  read_params(argv[1]);
  initialize_shoe(shoe, ndecks);
  shuffle(shoe, ndecks);

  ncards = 52*8;
  maxcardpos = floor(ncards*penet);

  startbank = bank;
  for(i = 0; i < ntrials; i++) {
    play();
  }
}

void play() {
  int curbets = 0, playeraction, flag = 0;
  bank = startbank;
  while ( !trialover(curbets) ) {
    handno = 0, nhands = 1;    
    opendraw();
    while( flag == 0 ) {
      playeraction = verb(1);      
      switch(playeraction) {
      case 0: // surrender
	bank += bets[handno--]/2.0;
	if (handno < 0) { flag = 1; } // Last hand surrendered
	else { recomputeptotal(); }
	break; 
      case 1: // stand
	if (--handno < 0) { flag = 2; } // Last hand stood
	else { recomputeptotal(); }	
	break;
      case 2: // hit
	hitdeal();
	if (handbust()) {
	  if (--handno < 0) { flag = 3; } // Last hand busted
	  else { recomputeptotal(); }	  
	}
	break;
      case 3: // double
	bank -= bets[handno];
	bets[handno] = 2*bets[handno];
	hitdeal();
	if (--handno < 0) { flag = 4; } // Last hand doubled
	else { recomputeptotal(); }	
	break;
      case 4: // split
	splitdeal(); // Not checking if split command is valid
	break;
      default: // do nothing
	break;
      }
    }
    resolvedeal();
  }
}

void resolvedeal() {

  // Check player busts and finish dealer card draws
  handno = nhands-1;
  
}

void splitdeal() {
  nhands++;
  if( cardpos >= maxcardpos - 2 ) {
    shuffle(shoe, ndecks);
    cardpos = 0;
  }
  bank -= bets[handno++];
  bets[handno] = bets[handno-1];
  player[handno+1][0] = player[handno++][1];
  player[handno-1][1] = shoe[cardpos++];
  player[handno][1] = shoe[cardpos++];
  cardno = 1;

  if (player[handno][0] == 1) {
    ptotal[handno] = 11; psoft = true; paces[handno]++;
  } else if (player[handno][0] > 10) {
    ptotal[handno] = 10;
  } else { ptotal[handno] = player[handno][0]; }

  if (player[handno][1] == 1) {
    ptotal[handno] += 11; psoft = true; paces[handno]++;
  } else if (player[handno][1] > 10) {
    ptotal[handno] += 10;
  } else { ptotal[handno] += player[handno][1]; }
  
}

void hitdeal() {
  if( cardpos >= maxcardpos ) {
    shuffle(shoe, ndecks);
    cardpos = 0;
  }
  player[handno][cardno++] = shoe[cardpos++];
  updateptotal();
}

void recomputeptotal() {

  cardno = 0; ptotal[handno] = 0;
  while( player[handno][cardno] != 0 ) {
    if (player[handno][cardno] == 1) {
      ptotal[handno] += 11; psoft = true;
      paces[handno]++;
    } else if (player[handno][cardno] > 10) {
      ptotal[handno] += 10;
    } else { ptotal[handno] += player[handno][cardno]; }
    cardno++;
  }
  cardno--;
  while(paces[handno] > 0) {
    ptotal[handno] -= 10; paces[handno]--;
    if (ptotal[handno] < 21) { break; }
  }
  if (paces[handno] == 0) { psoft = false; }

}

void updateptotal() {

  if (player[handno][cardno-1] == 1) {
    ptotal[handno] += 11; psoft = true; paces[handno]++;
  } else if (player[handno][cardno-1] > 10) {
    ptotal[handno] += 10;
  } else { ptotal[handno] += player[handno][cardno-1]; }

  if (ptotal[handno] > 21 ) {
    for( int ll = 0; ll < cardno; ll++) {
      if (player[handno][ll] == 1) { paces[handno]++; }
    }
    while(paces[handno] > 0) {
      ptotal[handno] -= 10; paces[handno]--;
      if (ptotal[handno] < 21) { break; }
    }
    if (paces[handno] == 0) { psoft = false; }
  }
  
}

void opendraw() {
  cleartable();
  if( cardpos >= maxcardpos - 4 ) {
    shuffle(shoe, ndecks);
    cardpos = 0;
  }
  bets[handno] = verb(0);
  bank -= bets[handno];
  // dealer[0] card is hidden from player
  dealer[0] = shoe[cardpos++];
  dealer[1] = shoe[cardpos++];
  player[handno][0] = shoe[cardpos++];
  player[handno][1] = shoe[cardpos++];
  cardno = 1;

  if (dealer[1] == 1) {
    dtotal = 11; dsoft = true; daces++;
  } else if (dealer[1] > 10) {
    dtotal = 10;
  } else { dtotal = dealer[1]; }
  
  if (player[handno][0] == 1) {
    ptotal[handno] = 11; psoft = true; paces[handno]++;
  } else if (player[handno][0] > 10) {
    ptotal[handno] = 10;
  } else { ptotal[handno] = player[handno][0]; }

  if (player[handno][1] == 1) {
    ptotal[handno] += 11; psoft = true; paces[handno]++;
  } else if (player[handno][1] > 10) {
    ptotal[handno] += 10;
  } else { ptotal[handno] += player[handno][1]; }
  
}

bool handbust() {
  if (ptotal[handno] > 21 && paces[handno] == 0) { return true; }
  return false;
}

int verb(int state) {
  // This is the player action function
  switch(state) {
  case 0: // pre-deal betting cycle, return bet amount
    return openbet();
  case 1: // cards dealt, return action
    return pdecision();
  default: // Same as case 0
    return openbet();
  }
}

int openbet() {
  switch(strategy) {
  case 0: // Test strategy
    return minbet;
  default: // Same as case 0
    return minbet;
  }
}

int pdecision() {
  /* return meanings
     0 == surrender
     1 == stand
     2 == hit
     3 == double
     4 == split
  */

  switch(strategy) {
  case 0: // Test strategy, hit if ptotal < 17, else stand
  default: // Same as case 0
    if (ptotal[handno] < 17) { return 2; }
    else { return 1; }
  }
 
}

void cleartable() {
  // Clean up hands
  for(int i = 0; i < 21; i++) {
    dealer[i] = 0;
    for(int j = 0; j < 8; j++) { player[j][i] = 0; }
  }
  for(int k = 0; k < 8; k++) {
    bets[k] = 0.0; ptotal[k] = 0; paces[k] = 0;
  }
  dtotal = 0; psoft = false; dsoft = false; daces = 0;
}

bool trialover(int curbetno) {
  if (nbets > 0) {
    if (curbetno > nbets || bank <= 0) {
      return true; } else {return false;}
  } else if (nbets < 0) {
    if (curbetno > abs(nbets)) { return true; }
    else { return false; }
  } else {
    if (bank <= 0) { return true; } else { return false; }
  }
}

void read_params(char* fname) {

  sprintf(mystring,"debug_trace");
  get_bool_param(fname, mystring, &debug_trace, true);
  sprintf(mystring,"ndecks");
  get_int_param(fname, mystring, &ndecks, debug_trace);
  if (ndecks < 1) { ndecks = 1; }
  if (ndecks > 8) { ndecks = 8; }
  sprintf(mystring,"ntrials");  
  get_int_param(fname, mystring, &ntrials, debug_trace);
  sprintf(mystring,"houserules");  
  get_int_param(fname, mystring, &houserules, debug_trace);
  sprintf(mystring,"strategy");  
  get_int_param(fname, mystring, &strategy, debug_trace);
  sprintf(mystring,"nbets");  
  get_int_param(fname, mystring, &nbets, debug_trace);
  sprintf(mystring,"penetration");  
  get_real_param(fname, mystring, &penet, debug_trace);
  if (penet < 2.0) { penet = 10.0; }
  if (penet > 100.0) { penet = 100.0; }  
  sprintf(mystring,"bank");  
  get_real_param(fname, mystring, &bank, debug_trace);
  sprintf(mystring,"minbet");  
  get_real_param(fname, mystring, &minbet, debug_trace);
  if (minbet > bank) { minbet = bank; }
  sprintf(mystring,"betspread");  
  get_real_param(fname, mystring, &betspread, debug_trace);  
  
}

void swap(int *a, int *b) {
  int temp = *a;
  *a = *b;
  *b = temp;
}


// Fischer-Yates random permutation of arr[]
// Move this to GPU eventually
void shuffle( int* arr, int ndeck ) {

  int ii, jj, kk;
  //  srand ( time(NULL) );

  // Shuffle thrice, just 'cause
  for (kk = 0; kk < 3; kk++) {
    // Start from the last element and swap one by one. We don't
    // need to run for the first element that's why ii > 0
    for (ii = ndeck*52-1; ii > 0; ii--){

      // Pick a random index from 0 to ii
      jj = rand() % (ii+1);

      // Swap arr[i] with the element at random index
      swap(&arr[ii], &arr[jj]);
    }
  }
}

void initialize_shoe(int* arr, int ndeck) {

  // 1s are aces, J = 11, Q = 12, K = 13
  for (int ii = 0; ii < ndeck; ii++) {
    for (int jj = 0; jj < 4; jj++) {    
      for (int kk = 0; kk < 13; kk++) {
	arr[ii*52+jj*13+kk] = kk+1;
      }
    }
  }

}
